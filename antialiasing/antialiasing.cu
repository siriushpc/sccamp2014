#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

// kernel 
__global__ void antialiasingDevice(int *mat, int a, int b,int *res)
{
  
  int sum = 0;
  int neig = 0;
  
  int j = blockIdx.x*blockDim.x + threadIdx.x;
  int i = blockIdx.y*blockDim.y + threadIdx.y;

  if((i < a) && (j < b)){
    for (int dx = -1; dx < 2; ++dx) 
      for (int dy = -1 ; dy < 2; ++dy) {
        int ni = i + dx;
        int nj = j + dy;
        if ((ni >= 0) && (ni < a) && (nj >= 0) && (nj < b)) {
          neig++;
          sum += mat[ni * b + nj];
        }
      }
    res[i * b + j] = sum / neig;
  }
}


int main(void)
{
  int *mat_h, *res_h;           // pointers to host memory
  int *mat_d, *res_d;                 // pointer to device memory
  hipError_t err;

  int a,b;
  scanf("%d %d", &a, &b);
  //cudaSetDevice(1);  
  // allocate arrays on host
  size_t size = a*b*sizeof(int);
  mat_h = (int *)malloc(size);
  res_h = (int *)malloc(size);
    
  for (int i = 0; i < a; ++i) {
  	for (int j = 0; j < b; ++j) {
  		scanf("%d", mat_h + (i * b + j));
  	}
  }

  // allocate array on device 
  err = hipMalloc((void **) &mat_d, size);
  if (err != hipSuccess)
    fprintf(stderr,"Problemas solicitando memoria para mat_d\n");
  err = hipMalloc((void **) &res_d, size); 
  if (err != hipSuccess)
    fprintf(stderr,"Problemas solicitando memoria para res_d\n");
  
  // copy data from host to device
  err = hipMemcpy(mat_d, mat_h, size, hipMemcpyHostToDevice); 
  if (err != hipSuccess)
    fprintf(stderr,"Problemas copiando memoria a device mat_d mat_h\n");

  float blockSize = 1024;
  dim3 dimBlock (ceil(b/blockSize), ceil(a/blockSize),1);
  dim3 dimGrid (blockSize, blockSize,1);
 
  float elapsed=0;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  antialiasingDevice <<< dimGrid, dimBlock >>> (mat_d, a, b, res_d);
  //cudaDeviceSynchronize();  

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsed, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  fprintf(stderr,"The elapsed time in gpu was %.8f ms\n", elapsed);

  // Retrieve result from device and store in b_h
  err = hipMemcpy(res_h, res_d, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess)
    fprintf(stderr,"Problemas copiando de device a host\n");

  //print results
  for (int i = 0; i < a; ++i) {
    for (int j = 0; j < b; ++j) {
    printf("%d ", res_h[i*b + j]);
    }
    printf("\n");
    }

  // cleanup
  free(mat_h); free(res_h); hipFree(res_d); hipFree(mat_d);
 
}
