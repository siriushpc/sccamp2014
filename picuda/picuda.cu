#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define CL 2000000LL

// kernel 
__global__ void picuda(double *res, long long W, hiprandState *states) {
  long long i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < W) {

    double ans = 0; 
    unsigned int seed = (unsigned int) (clock() * i);
    hiprand_init(seed, 0, 0, states + i);
    for (long long j = 0; j < CL; ++j) {
      //curandState s; // seed a random number generator 
      double x = hiprand_uniform_double(states + i); 
      double y = hiprand_uniform_double(states + i); 
      double bound = 1.0;
      ans += ((x*x + y*y) <= bound) ? 1 : 0;
    }
    res[i] = 4.0 * ans / (double) CL;
  }
}


int main(void) {
  double *res_h;           // pointers to host memory
  double *res_d;           // pointer to device memory
  hipError_t err;
  const long long W = 64048LL;
  size_t size = W*sizeof(double);
  res_h = (double *)malloc(size);


  err = hipMalloc((void **) &res_d, size); 
  if (err != hipSuccess)
    fprintf(stderr,"Problemas solicitando memoria para res_d\n");
  
  float blockSize = 1024;
  dim3 dimBlock (ceil(W/blockSize), 1, 1);
  dim3 dimGrid (blockSize, 1, 1);
  hiprandState *devStates; 
  hipMalloc( (void **)&devStates, W * sizeof(hiprandState) );
 
  picuda <<< dimGrid, dimBlock >>> (res_d, W, devStates);

  // Retrieve result from device and store in b_h
  err = hipMemcpy(res_h, res_d, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess)
    fprintf(stderr,"Problemas copiando de device a host\n");

  //print results
  double ans = 0;
  for (long long i = 0; i < W; ++i) {
    ans += res_h[i];
  }

  printf("Pi's value : %.10lf\n",  ans / W);
  // cleanup
  free(res_h);
  hipFree(res_d);

  return 0;
}
